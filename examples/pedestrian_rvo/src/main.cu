#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <string>


#include "flamegpu/flame_api.h"

#include "io.cuh"
#include "rvo_kernels.cuh"
#include "RVOGraph.cuh"

#define fg_getfloat(x) FLAMEGPU->getVariable<float>(#x)


FLAMEGPU_AGENT_FUNCTION(output_pedestrian_location, MsgNone, MsgSpatial3D) {
    // Output each agents publicly visible properties.
    FLAMEGPU->message_out.setVariable<int>("id", FLAMEGPU->getVariable<int>("id"));
//    FLAMEGPU->message_out.setLocation(
//        FLAMEGPU->getVariable<float>("x"),
//        FLAMEGPU->getVariable<float>("y"),
//        FLAMEGPU->getVariable<float>("z"));
     FLAMEGPU->message_out.setVariable<float>("x", FLAMEGPU->getVariable<float>("x"));
     FLAMEGPU->message_out.setVariable<float>("y", FLAMEGPU->getVariable<float>("y"));
     FLAMEGPU->message_out.setVariable<float>("z", FLAMEGPU->getVariable<float>("z"));
    FLAMEGPU->message_out.setVariable<float>("velx", FLAMEGPU->getVariable<float>("velx"));
    FLAMEGPU->message_out.setVariable<float>("vely", FLAMEGPU->getVariable<float>("vely"));

    return ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(move, MsgSpatial3D, MsgNone) {

    //Input
    RVOLine orcaLines[ORCA_ARRAY_SIZE];
    Ped neighbours[ORCA_ARRAY_SIZE];
    int orcaLineCount = 0;
    int neighbourCount = 0;
    float timeHorizonObst = 2.0f;
    float timeHorizon = 0.5f;
    float simTimeStep = FLAMEGPU->environment.get<float>("TIME_SCALER");
    float agentRadius = 0.15f;
    float agentMaxSpeed = 3.0f;
    float INTERACTION_RANGE  = FLAMEGPU->environment.get<float>("INTERACTION_RANGE");

    int id = FLAMEGPU->getVariable<int>("id");
    float2 agent_pos = make_float2(fg_getfloat(x), fg_getfloat(z));
    float2 agent_dest = make_float2(fg_getfloat(destx), fg_getfloat(desty));
    float2 agent_vel = make_float2(fg_getfloat(velx),fg_getfloat(vely));

    //Output
    float2 agentObsVector = make_float2(0, 0);
    int obsVectorCount = 0;
    float2 agent_newvel = make_float2(0, 0);

    ///Gets obstacles first
     getObstacles(orcaLines, orcaLineCount, timeHorizonObst, agent_pos, agent_vel, agentRadius, agentMaxSpeed,agentObsVector, obsVectorCount);
     int numObstLines = orcaLineCount;

    if (obsVectorCount > 0)
        agentObsVector = agentObsVector / ((float) obsVectorCount * 2.0f);

    for (const auto &message : FLAMEGPU->message_in(agent_pos.x, agent_pos.y, 0)){
        // Ignore self messages.
        if (message.getVariable<int>("id") != id) {
            const float message_x = message.getVariable<float>("x");
            const float message_y = message.getVariable<float>("y");
            const float message_z = message.getVariable<float>("z");
            const float message_velx = message.getVariable<float>("velx");
            const float message_vely = message.getVariable<float>("vely");
            float2 other_pos = make_float2(message_x, message_z); //Get position
            float2 other_vel = make_float2(message_velx, message_vely);
            float2 replusion_vec = agent_pos - other_pos;
            float separationSq = absSq(replusion_vec);

            if (separationSq > 0.001f && separationSq < INTERACTION_RANGE * INTERACTION_RANGE) {
                ///Need to sort agent by distance
                Ped p;
                p.pos = other_pos;
                p.vel = other_vel;
                p.separationSq = separationSq;
                addNeighbourSorted(neighbours, neighbourCount, p);

            }

        }

    }

    for (int i = 0;i < neighbourCount;i++) {
        addOrcaAgent(orcaLines, orcaLineCount, timeHorizon, simTimeStep, agent_pos, agent_vel, agentRadius,neighbours[i].pos, neighbours[i].vel, agentRadius);
    }

    float2 prefVel = agent_dest*agentMaxSpeed*0.5f;

    performCollisionAvoidance(orcaLines, orcaLineCount, numObstLines, agentMaxSpeed, prefVel, agent_newvel);

    agent_pos = agent_newvel*simTimeStep + agent_pos;

    FLAMEGPU->setVariable<float>("x", agent_pos.x);
    FLAMEGPU->setVariable<float>("z", agent_pos.y);
    FLAMEGPU->setVariable<float>("velx", agent_newvel.x);
    FLAMEGPU->setVariable<float>("vely", agent_newvel.y);

    return ALIVE;

}



int main(int argc, const char ** argv) {


    importSteerBenchXML("merseyrail.xml");


    /**
     * Environment boundaries
     * 
     */
     float envCommRadius = 5.0f;
     float3 envMin = make_float3(-50, -5, -50);
     float3 envMax = make_float3(50, 5, 50);
     float separationRadius = 0.5f;



    /**
     * Create pedestrian model
     * 
     */
    ModelDescription model("pedestrian_rvo");

    /**
     * GLOBALS
     */
    {
        EnvironmentDescription &env = model.Environment();

            
        env.add("TIME_SCALER", 0.02f);
        env.add("INTERACTION_RANGE", 5.0f);
        env.add("SEPARATION_RADIUS", separationRadius);
   
    }

    /**
     * Messages
     * 
     */
    {
        EnvironmentDescription &env = model.Environment();
        MsgSpatial3D::Description &message = model.newMessage<MsgSpatial3D>("pedestrian_location");
        // Set the range and bounds.
        message.setRadius(envCommRadius);
        message.setMin(envMin.x, envMin.y, envMin.z);
        message.setMax(envMax.x, envMax.y, envMax.z);

        // A message to hold the location of an agent.
        message.newVariable<int>("id");
        // X Y Z are implicit.
        // message.newVariable<float>("x");
        // message.newVariable<float>("y");
        // message.newVariable<float>("z");
        message.newVariable<float>("velx");
        message.newVariable<float>("vely");
    
        
        

    }

    /**
     * Agents
     * 
     */
    {
        AgentDescription &agent = model.newAgent("Pedestrian");
        agent.newVariable<int>("id");
        agent.newVariable<float>("x");
        agent.newVariable<float>("y");
        agent.newVariable<float>("z");
        agent.newVariable<float>("velx");
        agent.newVariable<float>("vely");
        agent.newVariable<float>("target_speed");
        agent.newVariable<int>("kill");

        //Navigation - a vector to the the destination
        agent.newVariable<float>("destx");
        agent.newVariable<float>("desty");

        agent.newFunction("output_pedestrian_location", output_pedestrian_location).setMessageOutput("pedestrian_location");
        agent.newFunction("move", move).setMessageInput("pedestrian_location");


    }

    /**
    * Control flow
    * 
    */
    {
        LayerDescription &layer = model.newLayer();
        layer.addAgentFunction(output_pedestrian_location);
    }
    {
        LayerDescription &layer = model.newLayer();
        layer.addAgentFunction(move);
    }

    /**
    * Model runner
    * 
    */
    CUDASimulation cuda_model(model);

    /**
     * Obstacles
     */
    std::vector<std::vector<float2>> obstacles;

    float subDiv = 2;
    for(int i = 0 ; i < subDiv; i++){
        for( int j = 0; j < subDiv; j++){
            float envWidth = envMax.x - envMin.x;
            float envHeight = envMax.z - envMin.z;
            float xSpace = envWidth/4.0f;
            float ySpace = envHeight/4.0f;
            float offx = i*envWidth*0.5f + xSpace + envMin.x;
            float offy = j*envHeight*0.5f + ySpace + envMin.z;
            float length = 10;
            std::vector<float2> obs = {
                    make_float2(offx,offy),
                    make_float2(offx+length,offy),
                    make_float2(offx+length,offy+length),
                    make_float2(offx,offy+length),
                    make_float2(offx,offy)
            };
            obstacles.push_back(obs);

        }
    }



    /**
    * Visualisation
    * 
    */
    ModelVis &visualisation = cuda_model.getVisualisation();
    {
        EnvironmentDescription &env = model.Environment();
        float envWidth = envMax.x - envMin.x;
        const float INIT_CAM = envMax.x * 1.25f;
        visualisation.setInitialCameraLocation(INIT_CAM, INIT_CAM, INIT_CAM);
        visualisation.setCameraSpeed(0.002f * envWidth);
        auto &ped_agt = visualisation.addAgent("Pedestrian");
        // Position vars are named x, y, z; so they are used by default
        ped_agt.setModel(Stock::Models::ICOSPHERE);
        ped_agt.setModelScale(separationRadius);

        //Env bounds
        {
            auto pen = visualisation.newLineSketch(1, 1, 1, 0.2f);  // white
            pen.addVertex(envMax.x, 0, envMax.z);
            pen.addVertex(envMin.x, 0, envMax.z);
            pen.addVertex(envMax.x, 0, envMin.z);
            pen.addVertex(envMin.x, 0, envMin.z);

            pen.addVertex(envMax.x, 0, envMax.z);
            pen.addVertex(envMax.x, 0, envMin.z);
            pen.addVertex(envMin.x, 0, envMax.z);
            pen.addVertex(envMin.x, 0, envMin.z);

            //Visualise obstacles
            for( auto& obs : obstacles){
                for( int i = 0 ; i < obs.size() -1; i++){
                    pen.addVertex(obs[i].x, 0, obs[i].y);
                    pen.addVertex(obs[i+1].x, 0, obs[i+1].y);
                }
            }


        }

    }
    visualisation.activate();


    // Initialisation
    cuda_model.initialise(argc, argv);

    // If no xml model file was is provided, generate a population.
    {
        int w = 10;
        int h = 10;
        float envWidth = envMax.x - envMin.x;
        float envHeight = envMax.z - envMin.z;
        float x_space = (envWidth*0.1f)/(float)w ;
        float y_space = (envHeight*0.1f)/(float)h;
        int populationSize = w*h;
        int id = 0;
        AgentPopulation population(model.Agent("Pedestrian"), populationSize);
        std::default_random_engine rng;
        std::uniform_real_distribution<float> dist(-1.0f, 1.0f);
        for(int i = 0; i < w; i++){
            for(int j = 0; j < h; j++){
                float x = (float)i*x_space;// - (envWidth*0.5f);
                float y = (float)j*y_space;// - (envHeight*0.5f);
                AgentInstance instance = population.getNextInstance();
                instance.setVariable<int>("id", id);
                instance.setVariable<float>("x", x);
                instance.setVariable<float>("y", 0);
                instance.setVariable<float>("z", y);
                instance.setVariable<float>("velx", 0);
                instance.setVariable<float>("vely", 0);
                // instance.setVariable<float>("destx", dist(0));
                // instance.setVariable<float>("desty", dist(0));
                instance.setVariable<float>("destx", dist(rng));
                instance.setVariable<float>("desty", dist(rng));
                id += 1;

            }
        }
        cuda_model.setPopulationData(population);

    }

    /**
     * Build RVO data structures
     *
     */
     auto rvoGraph = new RVOGraph();
     rvoGraph->buildRVO(obstacles, getRVOObstaclePointer(), getRVOKDNodePointer());




    /**
     * Execution
     */
    cuda_model.simulate();

    visualisation.join();

    return 0;

}