#include "hip/hip_runtime.h"
#include "io.cuh"
#include "tinyxml2/tinyxml2.h"
#include "flamegpu/exception/FGPUException.h"
#include <stdlib.h>
#include <random>

using namespace tinyxml2;
using namespace std;

std::random_device rd;
std::mt19937 e2(rd());
std::uniform_real_distribution<> dist(0, 1);

#ifndef XMLCheckResult
#define XMLCheckResult(a_eResult) if (a_eResult != tinyxml2::XML_SUCCESS) { FGPUException::setLocation(__FILE__, __LINE__);\
    switch (a_eResult) { \
    case tinyxml2::XML_ERROR_FILE_NOT_FOUND : \
    case tinyxml2::XML_ERROR_FILE_COULD_NOT_BE_OPENED : \
        throw InvalidInputFile("TinyXML error: File could not be opened.\n Error code: %d", a_eResult); \
    case tinyxml2::XML_ERROR_FILE_READ_ERROR : \
        throw InvalidInputFile("TinyXML error: File could not be read.\n Error code: %d", a_eResult); \
    case tinyxml2::XML_ERROR_PARSING_ELEMENT : \
    case tinyxml2::XML_ERROR_PARSING_ATTRIBUTE : \
    case tinyxml2::XML_ERROR_PARSING_TEXT : \
    case tinyxml2::XML_ERROR_PARSING_CDATA : \
    case tinyxml2::XML_ERROR_PARSING_COMMENT : \
    case tinyxml2::XML_ERROR_PARSING_DECLARATION : \
    case tinyxml2::XML_ERROR_PARSING_UNKNOWN : \
    case tinyxml2::XML_ERROR_PARSING : \
        throw TinyXMLError("TinyXML error: Error parsing file.\n Error code: %d", a_eResult); \
    case tinyxml2::XML_ERROR_EMPTY_DOCUMENT : \
        throw TinyXMLError("TinyXML error: XML_ERROR_EMPTY_DOCUMENT\n Error code: %d", a_eResult); \
    case tinyxml2::XML_ERROR_MISMATCHED_ELEMENT : \
        throw TinyXMLError("TinyXML error: XML_ERROR_MISMATCHED_ELEMENT\n Error code: %d", a_eResult); \
    case tinyxml2::XML_CAN_NOT_CONVERT_TEXT : \
        throw TinyXMLError("TinyXML error: XML_CAN_NOT_CONVERT_TEXT\n Error code: %d", a_eResult); \
    case tinyxml2::XML_NO_TEXT_NODE : \
        throw TinyXMLError("TinyXML error: XML_NO_TEXT_NODE\n Error code: %d", a_eResult); \
    case tinyxml2::XML_ELEMENT_DEPTH_EXCEEDED : \
        throw TinyXMLError("TinyXML error: XML_ELEMENT_DEPTH_EXCEEDED\n Error code: %d", a_eResult); \
    case tinyxml2::XML_ERROR_COUNT : \
        throw TinyXMLError("TinyXML error: XML_ERROR_COUNT\n Error code: %d", a_eResult); \
    case tinyxml2::XML_NO_ATTRIBUTE: \
        throw TinyXMLError("TinyXML error: XML_NO_ATTRIBUTE\n Error code: %d", a_eResult); \
    case tinyxml2::XML_WRONG_ATTRIBUTE_TYPE : \
        throw TinyXMLError("TinyXML error: XML_WRONG_ATTRIBUTE_TYPE\n Error code: %d", a_eResult); \
    default: \
        throw TinyXMLError("TinyXML error: Unrecognised error code\n Error code: %d", a_eResult); \
    } \
}
#endif

std::string getString(XMLElement* node)
{
    if(node)
    {
        return std::string(node->GetText());
    }
    else{
        return std::string();
    }
}

float getInt(XMLElement* node)
{
    if(node){
        return node->IntText(0);
    }else{
        return 0;
    }
}

float getFloat(XMLElement* node)
{
    if(node){
        return node->FloatText(0);
    }
    else{
        return 0;
    }
}

float3 getFloat3(XMLElement* node, bool* isRandom = nullptr)
{
    if(node){
        if(node->FirstChildElement("random"))
        {
            if(isRandom) *isRandom = true;
            //Make random
            return make_float3(dist(e2), dist(e2), dist(e2));
        }
        else{
            if(isRandom) *isRandom = false;
            float x = getFloat(node->FirstChildElement("x"));
            float y = getFloat(node->FirstChildElement("y"));
            float z = getFloat(node->FirstChildElement("z"));
            return make_float3(x,y,z);
        }
    }
    else{
        if(isRandom) *isRandom = false;
        return make_float3(0,0,0);
    }

}


/**
 * Get a Bounds object from a node where its children xmin, xmax, ymin, ymax, zmin, and zmax.
 * @param node
 * @return Bounds Bounds object
 */
Bounds getBounds(XMLElement * node)
{
    auto xmin = node->FirstChildElement("xmin")->FloatText(0);
    auto xmax = node->FirstChildElement("xmax")->FloatText(0);
    auto ymin = node->FirstChildElement("ymin")->FloatText(0);
    auto ymax = node->FirstChildElement("ymax")->FloatText(0);
    auto zmin = node->FirstChildElement("zmin")->FloatText(0);
    auto zmax = node->FirstChildElement("zmax")->FloatText(0);
    Bounds bounds;
    bounds.min = make_float3(xmin, ymin, zmin);
    bounds.max = make_float3(xmax, ymax, zmax);

    return bounds;
}

/**
 * Get a sequence of agent goals from a goalSequence xml element
 * @param goalSequenceNode A goalSequence xml element node
 * @param goalsVector A global vector of agent goals
 * @return int The starting index position in the goalsVector of the retrieved goal
 */
int getAgentGoals(XMLElement* goalSequenceNode, std::vector<AgentGoal> &goalsVector)
{
    int goalIndex = goalsVector.size();

    auto goalNode = goalSequenceNode->FirstChild();
    while(goalNode){

        int currentIndex = goalsVector.size();
        AgentGoal goal;

        //Get goal type
        std::string goalName = std::string(goalNode->Value());
        if(goalName.compare("seekDynamicTarget") == 0){
            goal.goalType = AGENT_GOAL_SEEK_TARGET;

        }
        else if(goalName.compare("fleeDynamicTarget") == 0){
            goal.goalType = AGENT_GOAL_FLEE_TARGET;

        }
        else if(goalName.compare("seekStaticTarget") == 0){
            goal.goalType = AGENT_GOAL_SEEK_TARGET;

        }
        else if(goalName.compare("fleeStaticTarget") == 0){
            goal.goalType = AGENT_GOAL_FLEE_TARGET;

        }
        else if(goalName.compare("idle") == 0){
            goal.goalType = AGENT_GOAL_IDLE;
        }


        goal.targetName = getString(goalNode->FirstChildElement("targetName"));
        goal.desiredSpeed = getFloat(goalNode->FirstChildElement("desiredSpeed"));
        goal.timeDuration = getFloat(goalNode->FirstChildElement("timeDuration"));
        goal.targetLocation = getFloat3(goalNode->FirstChildElement("targetLocation"));

        goalNode = goalNode->NextSibling();
        if(goalNode){
            goal.nextIndex = currentIndex + 1; //Has more goals
        }
        else{
            goal.nextIndex = -1; //End of goals list
        }

        goalsVector.push_back(goal);

    }

    return goalIndex;
}

/**
 * Parse a steersuite test case format xml into a SimulationSpec
 * @param filePath Path to the xml file
 * @return Shared pointer to a SimulationSpec that contains all information to create a test case
 */
SimulationSpecPtr importSteerBenchXML(string filePath)
{
    SimulationSpecPtr env(new SimulationSpec());

    XMLDocument doc;
    XMLError errorId = doc.LoadFile(filePath.c_str());
    XMLCheckResult(errorId);

    XMLNode* pRoot = doc.FirstChildElement("SteerBenchTestCase");
    if(pRoot == nullptr){
        THROW TinyXMLError("TinyXML error: Error parsing doc %s.", filePath.c_str());
    }

    auto headerElem = pRoot->FirstChildElement("header");
    if(!headerElem){
        THROW TinyXMLError("TinyXML error: No header element");
    }

    auto worldBounds = headerElem->FirstChildElement("worldBounds");
    if(!worldBounds){
        THROW TinyXMLError("TinyXML error: No worldBounds element");
    }
    env->envBounds = getBounds(worldBounds);
    //Gets the environment boundaries



    //Obstacles
    int obstacleCount = 0;
    auto obstacleNode = pRoot->FirstChildElement("obstacle");
    while(obstacleNode)
    {
        auto bounds = getBounds(obstacleNode);
        env->obstacles.push_back(bounds);
        obstacleCount ++;
        obstacleNode = obstacleNode->NextSiblingElement("obstacle");
    }

    //Obstacle regions
    auto obtacleRegionNode = pRoot->FirstChildElement("obstacleRegion");
    while(obtacleRegionNode)
    {
        ObstacleRegion obstacleRegion;
        obstacleRegion.numObstacles = getInt(obtacleRegionNode->FirstChildElement("numObstacles"));
        obstacleRegion.obstacleSize = getFloat(obtacleRegionNode->FirstChildElement("obstacleSize"));
        obstacleRegion.obstacleHeight = getFloat(obtacleRegionNode->FirstChildElement("numObstacles"));
        obstacleRegion.regionBounds = getBounds(obtacleRegionNode->FirstChildElement("obstacleHeight"));

        env->obstacleRegions.push_back(obstacleRegion);
        obtacleRegionNode = obtacleRegionNode->NextSiblingElement("obstacleRegion");
    }

    //Agents
    auto agentNode = pRoot->FirstChildElement("agent");
    while(agentNode)
    {
        Agent agent;

        auto agentNameNode = agentNode->FirstChildElement("name");
        if(agentNameNode)
            agent.name = std::string(agentNameNode->GetText());

        auto initialConditionsNode = agentNode->FirstChildElement("initialConditions");
        agent.radius = getFloat(initialConditionsNode->FirstChildElement("radius"));
        agent.position = getFloat3(initialConditionsNode->FirstChildElement("position"));
        agent.direction = getFloat3(initialConditionsNode->FirstChildElement("direction"));
        agent.speed = getFloat(initialConditionsNode->FirstChildElement("speed"));

        agent.goalIndex = getAgentGoals(agentNode->FirstChildElement("goalSequence"), env->agentGoals);

        env->agents.push_back(agent);

        agentNode = agentNode->NextSiblingElement("agent");

    }

    //Agent regions
    auto agentRegionNode = pRoot->FirstChildElement("agentRegion");
    while(agentRegionNode != nullptr)
    {
        Agent agent;

        agent.numAgents = getFloat(agentRegionNode->FirstChildElement("numAgents"));
        agent.regionBounds = getBounds(agentRegionNode->FirstChildElement("regionBounds"));

        auto initialConditionsNode = agentRegionNode->FirstChildElement("initialConditions");
        agent.radius = getFloat(initialConditionsNode->FirstChildElement("radius"));
        agent.position = getFloat3(initialConditionsNode->FirstChildElement("position"));
        agent.direction = getFloat3(initialConditionsNode->FirstChildElement("direction"), &agent.isDirectionRandom);
        agent.speed = getFloat(initialConditionsNode->FirstChildElement("speed"));

        agent.goalIndex = getAgentGoals(agentRegionNode->FirstChildElement("goalSequence"), env->agentGoals);

        env->agentRegions.push_back(agent);

        agentRegionNode = agentRegionNode->NextSiblingElement("agentRegion");

    }


    return env;
}

/**
 * AgentRegions and ObstacleRegions allows the declaration of multiple pedestrian agents or obstacles in single element.
 * This functions expands these regions, creating conrete agents and obstacles and adds them to the agents and obstacles
 * vector respectively.
 * @param env Spec that has agentRegion or obstacleRegion objects
 */
void expandSpecRegions(SimulationSpecPtr env)
{
    std::random_device rd;
    std::mt19937 e2(rd());

    // Expand obstacle regions
    for( auto& obstacleRegion: env->obstacleRegions)
    {
        //Always square obstacle, height not used
        auto obsWidth = obstacleRegion.obstacleSize;
        auto regionBounds = obstacleRegion.regionBounds;
        std::uniform_real_distribution<> distx(regionBounds.min.x, regionBounds.max.x - obsWidth);
//        std::uniform_real_distribution<> disty(regionBounds.min.y, regionBounds.max.y - obsWidth);
        std::uniform_real_distribution<> distz(regionBounds.min.z, regionBounds.max.z - obsWidth);
        for(int i = 0 ; i < obstacleRegion.numObstacles; i++){
            Bounds b;
            b.min = make_float3(distx(e2), 0, distz(e2));
            b.max = make_float3(b.min.x + obsWidth, 0, b.min.z + obsWidth);
            env->obstacles.push_back(b);
        }

    }

    //Expand agent regions
    for( auto& agentRegion: env->agentRegions)
    {
        auto regionBounds = agentRegion.regionBounds;
        std::uniform_real_distribution<> distx(regionBounds.min.x, regionBounds.max.x);
//        std::uniform_real_distribution<> disty(regionBounds.min.y, regionBounds.max.y - obsWidth);
        std::uniform_real_distribution<> distz(regionBounds.min.z, regionBounds.max.z);

        for( int i = 0; i < agentRegion.numAgents; i++)
        {
            Agent agent;
            agent.position = make_float3(distx(e2), 0, distz(e2));
            agent.radius = agentRegion.radius;
            agent.speed = agentRegion.speed;
            if(agentRegion.isDirectionRandom){
                agent.direction = make_float3(dist(e2), 0, dist(e2));
            }
            else{
                agent.direction = agentRegion.direction;
            }

            agent.goalIndex = agentRegion.goalIndex;

            env->agents.push_back(agent);
        }
    }


}

/**
 * Create a counter-clockwise line of float2 to represent the boundary
 * @param bounds
 * @return
 */
std::vector<float2> getLineFromBounds(Bounds& bounds){
    std::vector<float2> line;
    line.push_back(make_float2(bounds.min.x, bounds.min.z));
    line.push_back(make_float2(bounds.max.x, bounds.min.z));
    line.push_back(make_float2(bounds.max.x, bounds.max.z));
    line.push_back(make_float2(bounds.min.x, bounds.max.z));
    return line;
}

/**
 * Create an example test case (used for when spec xml is not defined)
 * @return
 */
SimulationSpecPtr createTestSimSpec(){
    SimulationSpecPtr simSpec(new SimulationSpec());
    //Create agents by default
    simSpec->envBounds.min = make_float3(-50, -5, -50);
    simSpec->envBounds.max = make_float3(50, 5, 50);

    //Create obstacles
    float subDiv = 2;
    for(int i = 0 ; i < subDiv; i++){
        for( int j = 0; j < subDiv; j++){
            float envWidth = simSpec->envBounds.max.x - simSpec->envBounds.min.x;
            float envHeight = simSpec->envBounds.max.z - simSpec->envBounds.min.z;
            float xSpace = envWidth/4.0f;
            float ySpace = envHeight/4.0f;
            float offx = i*envWidth*0.5f + xSpace + simSpec->envBounds.min.x;
            float offy = j*envHeight*0.5f + ySpace + simSpec->envBounds.min.z;
            float length = 10;
            Bounds obs;
            obs.min = make_float3(offx, 0, offy);
            obs.max = make_float3(offx + length, 0, offy + length);
            simSpec->obstacles.push_back(obs);
        }
    }


    //Create agent goals
    auto ag = AgentGoal();
    ag.goalType = AGENT_GOAL_FLEE_TARGET;
    ag.desiredSpeed = 2.0;
    ag.timeDuration = 3;
    ag.targetLocation = make_float3(25, 0, 0);
    ag.nextIndex = 1;

    auto ag1 = AgentGoal();
    ag1.goalType = AGENT_GOAL_IDLE;
    ag1.desiredSpeed = 2.0;
    ag1.timeDuration = 3;
    ag1.targetLocation = make_float3(25, 0, 0);
    ag1.nextIndex = 2;

    auto ag2 = AgentGoal();
    ag2.goalType = AGENT_GOAL_SEEK_TARGET;
    ag2.desiredSpeed = 2.0;
    ag2.timeDuration = 1000;
    ag2.targetLocation = make_float3(25, 0, 0);
    ag2.nextIndex = -1;

    simSpec->agentGoals.push_back(ag);
    simSpec->agentGoals.push_back(ag1);
    simSpec->agentGoals.push_back(ag2);

    //Create agents
    int w = 10;
    int h = 10;
    float envWidth = simSpec->envBounds.max.x - simSpec->envBounds.min.x;
    float envHeight = simSpec->envBounds.max.z - simSpec->envBounds.min.z;
    float x_space = (envWidth*0.1f)/(float)w ;
    float y_space = (envHeight*0.1f)/(float)h;

    std::default_random_engine rng;
    std::uniform_real_distribution<float> dist(-1.0f, 1.0f);

    for(int i = 0; i < w; i++){
        for(int j = 0; j < h; j++){
            Agent agent;
            float x = (float)i*x_space;
            float y = (float)j*y_space;
            agent.position = make_float3(x, 0, y);
            agent.radius = 0.3;
            agent.speed = 2.0f;
            agent.direction = make_float3(dist(rng), 0, dist(rng));
            agent.goalIndex = 0;
            simSpec->agents.push_back(agent);


        }
    }

    //Create agent regions
    Agent agentRegion;
    agentRegion.numAgents = 10;
    agentRegion.regionBounds.min = make_float3(0,0,0);
    agentRegion.regionBounds.max = make_float3(10,0,10);
    agentRegion.radius = 0.3;
    agentRegion.speed = 2.0f;
    agentRegion.direction = make_float3(dist(rng), 0, dist(rng));
    agentRegion.goalIndex = 0;

    simSpec->agentRegions.push_back(agentRegion);

    return simSpec;
}